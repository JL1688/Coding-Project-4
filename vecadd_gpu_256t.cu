//
// Code Source: https://developer.nvidia.com/blog/even-easier-introduction-cuda/
//
// Modified by: Jiqing Liu
// course #:656-01, assignment number: 2025_05_09: CP#4 CPU version of vector addition code
// 05/09/2025
// 1-sentence description here: implementation of vector addition code (CPU version)
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
// Input arguments:
// - n: The number of elements in arrays x and y
// - x: The first array of n elements
// - y: The second array of n elements (result vector).
__global__ 
void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 29; // 512M elements

    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 512M elements on the GPU
    add<<<1, 256>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;
}
